#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/count.h>
#include <thrust/scan.h>


#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"




void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

// LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
// Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

// Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov, float DOF, float aperature){
  int index = x + (y * resolution.x);
  
  
  glm::vec3 alpha, beta, midPix, horizScale, vertScale, pixel;
  alpha  = glm::cross(view, up);
  beta   = glm::cross(alpha, view);
  midPix = eye + view;

  vertScale  = glm::normalize(beta)  * glm::length(view) * tan(glm::radians( - fov.y)); //had to flip this (it was upside down)
  horizScale = glm::normalize(alpha) * glm::length(view) * tan(glm::radians(fov.x));
  
  //jitter the pixel
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(-0.5,0.5);
  thrust::uniform_real_distribution<float> u02(-0.01,0.01);

  
  pixel = midPix + horizScale * (float)((2.0 * (x + (float)u01(rng))/resolution.x) - 1.0) + vertScale * (float)((2.0 * (y + (float)u01(rng))/resolution.y) - 1.0);
  ray r;
  
  //COMMENT OUT FOR DOF
  r.origin = eye;
  r.direction = glm::normalize(pixel - eye);
  
  /*   //UNCOMMENT FOR DOF
  r.origin = pixel;
  float aperatureOffsetX = (float)u01(rng) * aperature;//for DOF
  float aperatureOffsetY = (float)u01(rng) * aperature;//for DOF
  glm::vec3 focalDirection = glm::normalize(pixel - eye);
  glm::vec3 focalPoint = eye + (focalDirection * DOF);//for depth of field
  r.origin = r.origin + horizScale * (aperatureOffsetX/resolution.x) + vertScale * (aperatureOffsetY/resolution.y);
  r.direction = glm::normalize(focalPoint - r.origin);
  */
  return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

//Initialize rays
__global__ void initializeRay(glm::vec2 resolution, float time, cameraData cam, rayState* rayList){
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  if((x<=resolution.x && y<=resolution.y)){
    ray thisRay = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov, cam.DOF, cam.APERATURE);
    rayList[index].RAY      = thisRay;
    rayList[index].isValid  = 1;
    rayList[index].color    = glm::vec3(1,1,1);
    rayList[index].photoIDX = index;
  }
}

__global__ void initializeLightPaths(float time, cameraData cam, rayState* lightrayList, int numLightpaths){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if((index <= numLightpaths))
	{
		ray thisRay;
		//HARDCODED AS A POINT LIGHT 
		lightrayList[index].RAY.origin = glm::vec3(0, 9, 0); 

		thrust::default_random_engine rng(hash(index * time));
		thrust::uniform_real_distribution<float> u01(-1,1);
		thrust::uniform_real_distribution<float> u02(-1,1);
		float random  = (float) u01(rng);
		float random2 = (float) u02(rng);

		lightrayList[index].RAY.direction = getRandomDirectionInSphere(random, random2, lightrayList[index].RAY.origin); 
		lightrayList[index].isValid = true;
		lightrayList[index].color = glm::vec3(3.0);
	}
}

///////////////////////////////////
//////////////////////////////////
// TODO: IMPLEMENT THIS FUNCTION/ 
//   raytraceRay() should take in a camera, image buffer, geometry, materials, and lights, 
//   and should trace a ray through the scene and write the resultant color to a pixel in the image buffer.
////////////////////////////////
///////////////////////////////
// Core raytracer kernel
/*
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int maxDepth, glm::vec3* colors,
                            staticGeom* geoms, int numberOfGeoms, material* materials, int numberOfMaterials, 
                            rayState* rayList, int currDepth, int* validRays, int length){
  //need to update for string compaction
  //int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  //int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  //int index = x + (y * resolution.x);
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index < length){
  //if((x<=resolution.x && y<=resolution.y)){
    if(rayList[index].isValid == 0){
      return;
    }
    if(currDepth >= maxDepth){//exceeded max depth
       //this contribution is black
      colors[rayList[index].photoIDX] = (colors[rayList[index].photoIDX] * (time - 1.0f)/time) + (glm::vec3(0,0,0) * 1.0f/time);
      rayList[index].isValid = 0;
      validRays[index] = 0;
      return;
    }
    //get variables
    ray thisRay     = rayList[index].RAY;
    glm::vec3 COLOR = rayList[index].color;
    //intersection checks:
    float distToIntersect = FLT_MAX;//infinite distance
    float tmpDist;
    glm::vec3 tmpIntersectPoint, tmpIntersectNormal, intersectPoint, intersectNormal;
    material mat;
    
    for(int i = 0; i < numberOfGeoms; i++){
      if (geoms[i].type == SPHERE){
        tmpDist = sphereIntersectionTest(geoms[i], thisRay, tmpIntersectPoint, tmpIntersectNormal);
      }else if (geoms[i].type == CUBE){
        tmpDist = boxIntersectionTest(   geoms[i], thisRay, tmpIntersectPoint, tmpIntersectNormal);
      }//insert triangles here for meshes
      if (tmpDist != -1 && tmpDist < distToIntersect){ //hit is new closest
        distToIntersect = tmpDist;
        intersectNormal = tmpIntersectNormal;
        intersectPoint  = tmpIntersectPoint;
        mat = materials[geoms[i].materialid];
      }
    }
    //Did I intersect anything?
    if(distToIntersect == FLT_MAX){//miss
      //this contribution is black
      colors[rayList[index].photoIDX] = (colors[rayList[index].photoIDX] * (time - 1.0f)/time) + (glm::vec3(0,0,0) * 1.0f/time);
      rayList[index].isValid = 0;
      validRays[index] = 0;
    }else if(mat.emittance > 0.001){  //is this a light source?
      COLOR = COLOR * (mat.color * mat.emittance);
      colors[rayList[index].photoIDX] = (colors[rayList[index].photoIDX] * (time - 1.0f)/time) + (COLOR * 1.0f/time);
      rayList[index].isValid = 0;
      validRays[index] = 0;
      return;
    }
    
    //update variables
    thrust::default_random_engine rng(hash(index * (time + currDepth)));
    thrust::uniform_real_distribution<float> u01(0,1);
    calculateBSDF(thisRay, intersectPoint, intersectNormal, COLOR, mat, (float) u01(rng) ,(float) u01(rng)); 
    //update struct
    rayList[index].RAY   = thisRay;
    rayList[index].color = COLOR;
  }
}
*/

//Build Eye Path
__global__ void buildEyePath(glm::vec2 resolution, float time, cameraData cam, int maxDepth, glm::vec3* colors,
                            staticGeom* geoms, int numberOfGeoms, material* materials, int numberOfMaterials, 
                            rayState* rayList, int currDepth, Path* eyePaths){
  // index into array is based off pixel position
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  if((x<=resolution.x && y<=resolution.y)){
    if(rayList[index].isValid == 0){
      eyePaths[index].vert[currDepth].isValid = 0;
      return;
    }
    //clear vertices
    eyePaths[index].vert[currDepth].position = glm::vec3(0,0,0);
    eyePaths[index].vert[currDepth].colorAcc = glm::vec3(0,0,0);
    eyePaths[index].vert[currDepth].isValid  = 1;
    eyePaths[index].vert[currDepth].hitLight = 0;
    
    
    //get variables
    ray thisRay     = rayList[index].RAY;
    glm::vec3 COLOR = rayList[index].color;

    //intersection checks:
    float distToIntersect = FLT_MAX;//infinite distance
    float tmpDist;
    glm::vec3 tmpIntersectPoint, tmpIntersectNormal, intersectPoint, intersectNormal;
    material mat;
    
    for(int i = 0; i < numberOfGeoms; i++){
      if (geoms[i].type == SPHERE){
        tmpDist = sphereIntersectionTest(geoms[i], thisRay, tmpIntersectPoint, tmpIntersectNormal);
      }else if (geoms[i].type == CUBE){
        tmpDist = boxIntersectionTest(   geoms[i], thisRay, tmpIntersectPoint, tmpIntersectNormal);
      }//insert triangles here for meshes
      if (tmpDist != -1 && tmpDist < distToIntersect){ //hit is new closest
        distToIntersect = tmpDist;
        intersectNormal = tmpIntersectNormal;
        intersectPoint  = tmpIntersectPoint;
        mat = materials[geoms[i].materialid];
      }
    }
    
    //Did I intersect anything?
    if(distToIntersect == FLT_MAX){//miss
      //colors[rayList[index].photoIDX] = (colors[rayList[index].photoIDX] * (time - 1.0f)/time) + (glm::vec3(0,0,0) * 1.0f/time); //UPDATE PIXEL COLOR
      eyePaths[index].vert[currDepth].isValid = 0;
      rayList[index].isValid = 0;
      return;
    }else if(mat.emittance > 0.001){  //is this a light source?
      COLOR = COLOR * (mat.color * mat.emittance);
      //colors[rayList[index].photoIDX] = (colors[rayList[index].photoIDX] * (time - 1.0f)/time) + (COLOR * 1.0f/time); // UPDATE PIXEL COLOR
      eyePaths[index].vert[currDepth].hitLight = 1;
      eyePaths[index].vert[currDepth].colorAcc = COLOR;
      eyePaths[index].vert[currDepth].isValid = 1;
      return;
    }
    
    //save intersection point to eyePath
    eyePaths[index].vert[currDepth].position = intersectPoint;
    
    //update variables
    thrust::default_random_engine rng(hash(index * (time + currDepth)));
    thrust::uniform_real_distribution<float> u01(0,1);
    calculateBSDF(thisRay, intersectPoint, intersectNormal, COLOR, mat, (float) u01(rng) ,(float) u01(rng)); 

    //update struct
    rayList[index].RAY   = thisRay;
    rayList[index].color = COLOR;
    
    //save color to eyePath
    eyePaths[index].vert[currDepth].colorAcc = COLOR;
    eyePaths[index].vert[currDepth].isValid = 1;
  }
}

__global__ void connectPaths(glm::vec2 resolution, glm::vec3* colors, float* imageWeights, staticGeom* geoms, int numberOfGeoms, int traceDepth, Path* eyePaths, Path* lightPaths){
  // index into array is based off pixel position
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  if((x<=resolution.x && y<=resolution.y)){
    
    //updates all eye paths that hit a light source
    for (int lightIDX = 0; lightIDX < 10; lightIDX++){
      for (int idx = 0; idx < traceDepth; idx++){//traceDepth - 4; // First bounce of light
        for(int eyeVert = 0; eyeVert < traceDepth; eyeVert++){
          if (eyePaths[index].vert[eyeVert].isValid != 0 && lightPaths[lightIDX].vert[idx].isValid != 0){
            ray r; 
            r.origin = eyePaths[index].vert[eyeVert].position; 
            r.direction = glm::normalize(lightPaths[lightIDX].vert[idx].position - eyePaths[index].vert[eyeVert].position);
            //check intersection of this ray with scene
            float dist = glm::distance(lightPaths[lightIDX].vert[idx].position, r.origin);
            float distToIntersect = dist; //FLT_MAX;//infinite distance
            float tmpDist;
            glm::vec3 tmpIntersectPoint, tmpIntersectNormal;
    
            for(int i = 0; i < numberOfGeoms; i++){
            	if (geoms[i].type == SPHERE){
            		tmpDist = sphereIntersectionTest(geoms[i], r, tmpIntersectPoint, tmpIntersectNormal);
            	}else if (geoms[i].type == CUBE){
            		tmpDist = boxIntersectionTest(   geoms[i], r, tmpIntersectPoint, tmpIntersectNormal);
            	}//insert triangles here for meshes  //TODO: ADD MESH STUFF
            //update distance
              if (tmpDist != -1 && tmpDist < distToIntersect){ //hit is new closest
                distToIntersect = tmpDist;
              }
            }
            
            if(distToIntersect == dist){ //no intersection, we can add color
            	 //change weight calculation when we add other materials
            	float weight = imageWeights[index];
            	float denom  = weight + 1.0f;
            	glm::vec3 pathColor = eyePaths[index].vert[eyeVert].colorAcc  * lightPaths[lightIDX].vert[idx].colorAcc;
            	colors[index] = colors[index] * (weight/denom) + pathColor * (1.0f /denom);
            	imageWeights[index] = denom;
          //return;
            }
          }
        }
      }
    }
  }
}


__global__ void MISRenderColor(glm::vec2 resolution, glm::vec3* colors, float* imageWeights, int traceDepth, Path* eyePaths) {
  // index into array is based off pixel position
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  //integrate light contribution Back to Front.
  if(x<=resolution.x && y<=resolution.y){
    for(int vert = traceDepth - 1; vert >= 0; vert--){
      if(eyePaths[index].vert[vert].isValid){
        float weight = imageWeights[index];
        float denom  = weight + (float)(vert + 1.0f);
        colors[index] = colors[index] * (weight/denom) + eyePaths[index].vert[vert].colorAcc * (1.0f /denom);
        imageWeights[index] = denom;
        return;
      }
    }
  }
}


__global__ void compactRays(int* scanRays, rayState* rayList, int* validRays, int length){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index >= length){
    return;
  }
  validRays[index] = 0;
  if(index == 0){//first 
    return;
  }

  if(scanRays[index - 1] < scanRays[index]){
    rayState newRay = rayList[index];
    __syncthreads();
    rayList[scanRays[index]] = newRay;
    validRays[scanRays[index]] = 1;
  }
}


// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms, geom* lights, int numberOfLights){
  
  int traceDepth = 4; //determines how many bounces the raytracer traces

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  // send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage,           (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  // allocate eye path per pixel
  Path* eyePaths = NULL;
  hipMalloc((void**)&eyePaths,           (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(Path));

  // allocate Light paths
	Path* lightPaths = NULL;
	hipMalloc((void**)&lightPaths,         10 * sizeof(Path));
  
  // Allocate per-pixel accumulated weight (probabilites of valid light paths)
  float* imageWeights = NULL;
  hipMalloc((void**)&imageWeights,                  (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(float));
  hipMemcpy( imageWeights, renderCam->imageWeights, (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(float), hipMemcpyHostToDevice);
  
  // package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;
  }
  
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms,   numberOfGeoms * sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms * sizeof(staticGeom), hipMemcpyHostToDevice);
  
  
  // package Lights and sent to GPU
  staticGeom* lightList = new staticGeom[numberOfLights];
  for(int i=0; i<numberOfLights; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = lights[i].type;
    newStaticGeom.materialid = lights[i].materialid;
    newStaticGeom.translation = lights[i].translations[frame];
    newStaticGeom.rotation = lights[i].rotations[frame];
    newStaticGeom.scale = lights[i].scales[frame];
    newStaticGeom.transform = lights[i].transforms[frame];
    newStaticGeom.inverseTransform = lights[i].inverseTransforms[frame];
    lightList[i] = newStaticGeom;
  }
  
  staticGeom* cudalights = NULL;
  hipMalloc((void**)&cudalights,    numberOfLights * sizeof(staticGeom));
  hipMemcpy( cudalights, lightList, numberOfLights * sizeof(staticGeom), hipMemcpyHostToDevice);
  
  
  // package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;
  cam.DOF = renderCam->DOF[frame];//new
  cam.APERATURE = renderCam->APERATURE[frame];//new
  
  // package materials
  material* materialList = NULL;
  hipMalloc((void**) &materialList,   numberOfMaterials * sizeof(material));
  hipMemcpy( materialList, materials, numberOfMaterials * sizeof(material), hipMemcpyHostToDevice);
  
  //allocate Rays
  rayState* rayList = NULL;
  hipMalloc((void**)&rayList, (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(rayState));

  //allocate light rays 
  rayState* lightrayList = NULL;
  hipMalloc((void**)&lightrayList, 10 * sizeof(rayState));

  


  // kernel launches
  //Get initial rays
  initializeRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, rayList);

  //Initialize light subpaths
  int numLightpaths = 10;

  initializeLightPaths<<<1, 10>>>((float)iterations, cam, lightrayList, numLightpaths);
  

  //build eye path
  for(int i = 0; i < traceDepth; i++){
    //do one step
    buildEyePath<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms, materialList, numberOfMaterials, rayList, i, eyePaths);
  }

/*
   //buildLightPath
  for(int i = 0; i < traceDepth; i++){
    //do one step
    buildEyePath<<<1, numLightpaths>>>(glm::vec2(10,1), (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms, materialList, numberOfMaterials, lightrayList, i, lightPaths);
  }
*/


/*  
  //connect paths and render to screen
  connectPaths<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, cudaimage, imageWeights, cudageoms, numberOfGeoms, traceDepth, eyePaths, lightPaths);
*/

MISRenderColor<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, cudaimage, imageWeights, traceDepth, eyePaths);




// original Path Tracing Algorithm
  /*
   // kernel launches
  //Get initial rays
  initializeRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, rayList);
  thrust::device_vector<int> validRays((int)renderCam->resolution.x * (int)renderCam->resolution.y, 1);
  int* thrustArray = thrust::raw_pointer_cast( &validRays[0] );
  int length = thrust::count(validRays.begin(), validRays.end(), 1);//count valid rays
  thrust::device_vector<int> scanRay((int)renderCam->resolution.x * (int)renderCam->resolution.y, 0);
  int* scanPointer = thrust::raw_pointer_cast( &scanRay[0] );
  
  //depth trace with compaction
  for(int i = 0; i <= traceDepth; i++){
    //do one step
    raytraceRay<<<(int)ceil((float)length/64.0f), 64>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms, materialList, numberOfMaterials, rayList, i, thrustArray, length);
    //build scan
    thrust::exclusive_scan(validRays.begin(), validRays.end(), &scanRay[0]);
    scanPointer = thrust::raw_pointer_cast( &scanRay[0] );
    //compact rays
    compactRays<<<(int)ceil((float)length/64.0f), 64 >>>(scanPointer, rayList, thrustArray, length);
    //update length
    length = thrust::count(validRays.begin(), validRays.end(), 1);//count valid rays
  }
  */

  //update visual
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);

  // retrieve image from GPU
  hipMemcpy( renderCam->image,        cudaimage,    (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);
  //retrieve weights from GPU
  hipMemcpy( renderCam->imageWeights, imageWeights, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(float), hipMemcpyDeviceToHost);

  // free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  hipFree( cudalights ); //added
  hipFree(materialList); //added
  hipFree(rayList);      //added
  hipFree(lightrayList); //VCM added
  hipFree(eyePaths);     //added
  hipFree(lightPaths);     //added
  hipFree(imageWeights); //added
  delete geomList;
  delete lightList;//ADDED

  // make certain the kernel has completed
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}

