#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/count.h>
#include <thrust/scan.h>


#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"




void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

// LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
// Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

// Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov, float DOF, float aperature){
  int index = x + (y * resolution.x);
  
  
  glm::vec3 alpha, beta, midPix, horizScale, vertScale, pixel;
  alpha  = glm::cross(view, up);
  beta   = glm::cross(alpha, view);
  midPix = eye + view;

  vertScale  = glm::normalize(beta)  * glm::length(view) * tan(glm::radians( - fov.y)); //had to flip this (it was upside down)
  horizScale = glm::normalize(alpha) * glm::length(view) * tan(glm::radians(fov.x));
  
  //jitter the pixel
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(-0.5,0.5);
  thrust::uniform_real_distribution<float> u02(-0.01,0.01);

  
  pixel = midPix + horizScale * (float)((2.0 * (x + (float)u01(rng))/resolution.x) - 1.0) + vertScale * (float)((2.0 * (y + (float)u01(rng))/resolution.y) - 1.0);
  ray r;
  
  //COMMENT OUT FOR DOF
  r.origin = eye;
  r.direction = glm::normalize(pixel - eye);
  
  /*   //UNCOMMENT FOR DOF
  r.origin = pixel;
  float aperatureOffsetX = (float)u01(rng) * aperature;//for DOF
  float aperatureOffsetY = (float)u01(rng) * aperature;//for DOF
  glm::vec3 focalDirection = glm::normalize(pixel - eye);
  glm::vec3 focalPoint = eye + (focalDirection * DOF);//for depth of field
  r.origin = r.origin + horizScale * (aperatureOffsetX/resolution.x) + vertScale * (aperatureOffsetY/resolution.y);
  r.direction = glm::normalize(focalPoint - r.origin);
  */
  return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

//Initialize rays
__global__ void initializeRay(glm::vec2 resolution, float time, cameraData cam, rayState* rayList){
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  if((x<=resolution.x && y<=resolution.y)){
    ray thisRay = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov, cam.DOF, cam.APERATURE);
    rayList[index].RAY      = thisRay;
    rayList[index].isValid  = 1;
    rayList[index].color    = glm::vec3(1,1,1);
    rayList[index].photoIDX = index;
  }
}

__global__ void initializeLightPaths(float time, cameraData cam, rayState* lightrayList, int numLightpaths, staticGeom* lights, int numLights, Path* lightPaths, material* materialList){
	/*
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * cam.resolution.x);

	if((x<=cam.resolution.x && y<=cam.resolution.y))
	*/
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if((index <= numLightpaths))
	{
		//generate random numbers
		thrust::default_random_engine rng(hash(index * time));
		thrust::uniform_real_distribution<float> u01(-1,1);
		thrust::uniform_real_distribution<float> u02(0,1);
		float random  = (float) u02(rng) * 2.0 - 1.0;
		float random2 = (float) u02(rng) * 2.0 - 1.0;

		//randomly select light
		staticGeom light = lights[0];


		ray thisRay;
		glm::vec3 normal = getRandomDirectionInSphere(random, random2, glm::vec3(0,0,0)); // random point on unit sphere
		thisRay.origin = multiplyMV(light.transform, glm::vec4(normal,1.0f));
		random  = (float) u02(rng);
		random2 = (float) u02(rng);
		thisRay.direction = calculateRandomDirectionInHemisphere(normal, random, random2);

		//lightrayList[index].color = glm::vec3(3.0);
		lightrayList[index].RAY = thisRay;
		lightrayList[index].isValid = 1;

		//initialize light Path
		vertex v;
		v.isValid = 1;//This is the light!
		v.hitLight = 1;
		v.mat = materialList[light.materialid];
		v.outDirection = glm::vec3(0);
		v.inDirection = - thisRay.direction;
		v.pdfWeight = 1.0f;
		v.position = thisRay.origin;
		lightPaths[index].vert[0] = v;

	}
}

__host__ __device__ float getSolidAngle(staticGeom light, glm::vec3 position, glm::vec3 normal){
	
	glm::vec3 p1 = glm::normalize(glm::vec3(1,1,1)); //point on a unit sphere
	glm::vec3 pOnSphere = multiplyMV(light.transform, glm::vec4(p1,1.0f));//point on our sphere
	glm::vec3 centerOfSphere = multiplyMV(light.transform, glm::vec4(0,0,0,1.0f));// center of sphere
	
	glm::vec3 direction = centerOfSphere - position; 
	
	float radius = glm::distance(pOnSphere, centerOfSphere);
	float dist = glm::length(direction); 
	float angle = glm::atan(radius/dist);
	float solid = TWO_PI * (1.0f - glm::cos(angle));
	return solid *(1.0/PI);
	//Convert to PDFWeight
	//return solid * (dist * dist) / abs(glm::dot(normal,direction))/TWO_PI;
}

__host__ __device__ float convertSolidAngle(float value, float dist, float cos){
	return value * dist * dist / abs(cos); 

}
__host__ __device__ float intersectionTest(staticGeom* geoms, int numberOfGeoms, int& materialID, ray thisRay, float distToIntersect, glm::vec3& intersectNormal, glm::vec3& intersectPoint){
  float tmpDist;
  glm::vec3 tmpIntersectPoint, tmpIntersectNormal;
  for(int i = 0; i < numberOfGeoms; i++){
    staticGeom geometry = geoms[i];
    if (geometry.type == SPHERE){
      tmpDist = sphereIntersectionTest(geometry, thisRay, tmpIntersectPoint, tmpIntersectNormal);
    }else if (geometry.type == CUBE){
      tmpDist = boxIntersectionTest(   geometry, thisRay, tmpIntersectPoint, tmpIntersectNormal);
    }//insert triangles here for meshes
    if (tmpDist != -1 && tmpDist < distToIntersect){ //hit is new closest
      materialID = geometry.materialid;//index of hit material
      distToIntersect = tmpDist;
      intersectNormal = tmpIntersectNormal;
      intersectPoint  = tmpIntersectPoint;
    }
  }
  return distToIntersect;
}

__host__ __device__ glm::vec3 directLightContribution(material m, staticGeom* geoms, int numberOfGeoms, staticGeom* lights, int numberOfLights, 
	material* materials, glm::vec3 normal, glm::vec3 inDirection, glm::vec3 intersectionPoint, float rnd1, float rnd2, float& solidAngle){
  /////////////////////////////////////////////////
  //TODO: Update to support multiple light sources
  //  - Currently assumes all lights are spheres
  ////////////////////////////////////////////////
/*
  if(m.type == 1){
	  solidAngle = 0.0f;
	  return glm::vec3(0);

  }
*/
  
  //Get random point on light
  glm::vec3 lightPOS = getLightPos(lights, rnd1, rnd2); 
  float dist = glm::distance(lightPOS, intersectionPoint);
  
  //make ray
  ray thisRay;
  thisRay.origin = intersectionPoint;
  thisRay.direction = glm::normalize(lightPOS - thisRay.origin);
  //intersection checks
  float distToIntersect = dist; //distance to light intersection
  int materialID;                            //updated in intersectionTest
  glm::vec3 intersectPoint, intersectNormal; //updated in intersectionTest
  distToIntersect = intersectionTest(geoms, numberOfGeoms, materialID, thisRay, distToIntersect, intersectNormal, intersectPoint);
  
  glm::vec3 dirColor;
  if(distToIntersect < dist && materialID != 9){//in shadow
    dirColor = glm::vec3(0,0,0);
  }else{
    material lightMaterial = materials[lights[0].materialid];
    glm::vec3 lightColor = lightMaterial.color * lightMaterial.emittance;
    ///////////////////////////////
    //MODIFY THIS FOR OTHER BSDFS
    //////////////////////////////
    dirColor = getColorFromBSDF(inDirection, thisRay.direction, normal, lightColor, m);
  }
  //calculate solid angle
  solidAngle = getSolidAngle(lights[0], intersectionPoint, normal);
  return dirColor;
}

//Build Eye Path
__global__ void buildEyePath(glm::vec2 resolution, float time, cameraData cam, int maxDepth, glm::vec3* colors,
                            staticGeom* geoms, int numberOfGeoms, material* materials, int numberOfMaterials, 
                            rayState* rayList, int currDepth, Path* eyePaths, staticGeom* lights, int numberOfLights){
  // index into array is based off pixel position
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  if((x<=resolution.x && y<=resolution.y)){
    rayState rs = rayList[index];
    if(rs.isValid == 0){
      eyePaths[index].vert[currDepth].isValid = 0;
      return;
    }
    //clear vertices
    vertex v;
    v.isValid  = 1;
    v.hitLight = 0;
    
    
    //random number generator
    thrust::default_random_engine rng(hash(index * (time + currDepth)));
    thrust::uniform_real_distribution<float> u01(0,1);
    
    //get variables
    ray thisRay     = rs.RAY;
    glm::vec3 COLOR = rs.color;
    v.inDirection = thisRay.direction;
    
    //intersection checks:
    float distToIntersect = FLT_MAX;
    int materialID = 0;                        //updated in intersectionTest
    glm::vec3 intersectPoint, intersectNormal; //updated in intersectionTest
    distToIntersect = intersectionTest(geoms, numberOfGeoms, materialID, thisRay, distToIntersect, intersectNormal, intersectPoint);
    material mat = materials[materialID];
    
    float solidAngle;
    glm::vec3 directLight;
    
    if(distToIntersect == FLT_MAX){//miss
      v.isValid = 0;
      eyePaths[index].vert[currDepth] = v; //update invalid vertex!
      rayList[index].isValid = 0;
      return;
    }else if(mat.type == 9){  //is this a light source?
      solidAngle = 1.0;//TWO_PI;//noDirectLight
      directLight = (mat.color * mat.emittance);
      COLOR = COLOR * directLight;
      v.hitLight = 1;
      v.isValid = 1;
      rayList[index].isValid = 0;
    }else{
	  solidAngle = 0.0;
      directLight = directLightContribution(mat, geoms, numberOfGeoms, lights, numberOfLights, materials, intersectNormal, thisRay.direction, intersectPoint, (float) u01(rng) ,(float) u01(rng), solidAngle); //updates solidAngle as side effect
    }


    //save intersection point to eyePath
    v.position = intersectPoint;
    
    //update variables
    float pdfWeight = 0;
    calculateBSDF(thisRay, intersectPoint, intersectNormal, COLOR, mat, (float) u01(rng) ,(float) u01(rng), pdfWeight, lights); 

    //update rayList
    rs.RAY   = thisRay;
    rs.color = COLOR;
    rayList[index] = rs;
    
    //save color to eyePath
    v.colorAcc = COLOR; //Saves color at each vertex although i think we only need the last one???
    v.normal = intersectNormal;
    v.outDirection = thisRay.direction; 
    v.mat = mat;

    //update PathProbability
    if(currDepth == 0){
      v.pathProbability = pdfWeight;
    }else{
      v.pathProbability = eyePaths[index].vert[currDepth - 1].pathProbability * pdfWeight; //Update Path Weight
    }
    v.directLight = directLight;
    v.solidAngle = solidAngle;//convertSolidAngle(solidAngle, distToIntersect, glm::dot(intersectNormal, thisRay.direction)) * solidAngle;
    v.pdfWeight = pdfWeight;  //probability of this bounce only
    eyePaths[index].vert[currDepth] = v;
  }
}

//Build Light Path
__global__ void buildLightPath(glm::vec2 resolution, float time, cameraData cam, int maxDepth, glm::vec3* colors,
                            staticGeom* geoms, int numberOfGeoms, material* materials, int numberOfMaterials,
                            rayState* rayList, int currDepth, Path* lightPaths, staticGeom* lights, int numberOfLights){
  // index into array is based off pixel position
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  if((x<=resolution.x && y<=resolution.y)){
    rayState rs = rayList[index];
    if(rs.isValid == 0){
      lightPaths[index].vert[currDepth].isValid = 0;
      return;
    }
    //clear vertices
    vertex v;
    v.isValid  = 1;
    v.hitLight = 0;


    //random number generator
    thrust::default_random_engine rng(hash(index * (time + currDepth)));
    thrust::uniform_real_distribution<float> u01(0,1);

    //get variables
    ray thisRay     = rs.RAY;
    glm::vec3 COLOR = rs.color;
    v.outDirection =  - thisRay.direction; //in light paths we're going backward

    //intersection checks:
    float distToIntersect = FLT_MAX;
    int materialID = 0;                        //updated in intersectionTest
    glm::vec3 intersectPoint, intersectNormal; //updated in intersectionTest
    distToIntersect = intersectionTest(geoms, numberOfGeoms, materialID, thisRay, distToIntersect, intersectNormal, intersectPoint);
    material mat = materials[materialID];

    float solidAngle;
    glm::vec3 directLight;

    if(distToIntersect == FLT_MAX){//miss
      v.isValid = 0;
      lightPaths[index].vert[currDepth] = v; //update invalid vertex!
      rayList[index].isValid = 0;
      return;
    }else if(mat.type == 9){  //is this a light source?
      solidAngle = 1.0f;//TWO_PI;//noDirectLight
      directLight = (mat.color * mat.emittance);
      COLOR = COLOR * directLight;
      v.hitLight = 1;
      v.isValid = 0;
      rayList[index].isValid = 0; //I could probably let it continue...
    }else{
	  solidAngle = 0.0;
      directLight = directLightContribution(mat, geoms, numberOfGeoms, lights, numberOfLights, materials, intersectNormal, thisRay.direction, intersectPoint, (float) u01(rng) ,(float) u01(rng), solidAngle); //updates solidAngle as side effect
    }


    //save intersection point to lightPath
    v.position = intersectPoint;

    //update variables
    float pdfWeight = 0;
    calculateBSDF(thisRay, intersectPoint, intersectNormal, COLOR, mat, (float) u01(rng) ,(float) u01(rng), pdfWeight, lights);

    //update rayList
    rs.RAY   = thisRay;
    rs.color = COLOR;
    rayList[index] = rs;

    //save color to eyePath
    v.colorAcc = COLOR; //Saves color at each vertex although i think we only need the last one???
    v.normal = intersectNormal;
    v.inDirection = - thisRay.direction; // we're tracing backward
    v.mat = mat;

    //update PathProbability
    //start at depth 1 not 0;
    v.pathProbability = lightPaths[index].vert[currDepth - 1].pathProbability * pdfWeight; //Update Path Weight

    v.directLight = directLight;
    v.solidAngle = solidAngle;//convertSolidAngle(solidAngle, distToIntersect, glm::dot(intersectNormal, thisRay.direction)) * solidAngle;
    v.pdfWeight = pdfWeight;  //probability of this bounce only
    lightPaths[index].vert[currDepth] = v;
  }
}


__global__ void connectPaths(glm::vec2 resolution, glm::vec3* colors, float* imageWeights, staticGeom* geoms, int numberOfGeoms, int traceDepth, Path* eyePaths, Path* lightPaths){
  // index into array is based off pixel position
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  if((x<=resolution.x && y<=resolution.y)){
    
    //updates all eye paths that hit a light source
    for (int lightIDX = 0; lightIDX < 10; lightIDX++){
      for (int idx = 0; idx < traceDepth; idx++){//traceDepth - 4; // First bounce of light
        for(int eyeVert = 0; eyeVert < traceDepth; eyeVert++){
          if (eyePaths[index].vert[eyeVert].isValid != 0 && lightPaths[lightIDX].vert[idx].isValid != 0){
            ray thisRay; 
            thisRay.origin = eyePaths[index].vert[eyeVert].position; 
            thisRay.direction = glm::normalize(lightPaths[lightIDX].vert[idx].position - eyePaths[index].vert[eyeVert].position);
            //check intersection of this ray with scene
            float dist = glm::distance(lightPaths[lightIDX].vert[idx].position, thisRay.origin);
            float distToIntersect = dist;
            int materialID = 0;                        //updated in intersectionTest
            glm::vec3 intersectPoint, intersectNormal; //updated in intersectionTest
            distToIntersect = intersectionTest(geoms, numberOfGeoms, materialID, thisRay, distToIntersect, intersectNormal, intersectPoint);
            
            if(distToIntersect == dist){ //no intersection, we can add color
            	 //change weight calculation when we add other materials
            	float weight = imageWeights[index];
            	float denom  = weight + 1.0f;
            	glm::vec3 pathColor = eyePaths[index].vert[eyeVert].colorAcc  * lightPaths[lightIDX].vert[idx].colorAcc;
            	colors[index] = colors[index] * (weight/denom) + pathColor * (1.0f /denom);
            	imageWeights[index] = denom;
          //return;
            }
          }
        }
      }
    }
  }
}
//NO DIRECT LIGHTING CONTRIBUTION
__global__ void RenderColor(glm::vec2 resolution, glm::vec3* colors, float* imageWeights, int traceDepth, Path* eyePaths) {
  // index into array is based off pixel position
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  //integrate light contribution Back to Front.
  if(x<=resolution.x && y<=resolution.y){
    for(int vert = traceDepth - 1; vert >= 0; vert--){
      if(eyePaths[index].vert[vert].isValid == 1 && eyePaths[index].vert[vert].hitLight == 1){
        float weight = imageWeights[index];
        float pdfWeight = eyePaths[index].vert[vert].pathProbability;
        float denom  = weight + pdfWeight;
        colors[index] = colors[index] * (weight/denom) + eyePaths[index].vert[vert].colorAcc * (pdfWeight /denom);
        imageWeights[index] = denom;
        return;
      }
    }
  }
}

//Only render direct lighting on first bounce
__global__ void RenderDirectLight(glm::vec2 resolution, glm::vec3* colors, float* imageWeights, int traceDepth, Path* eyePaths) {
  // index into array is based off pixel position
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  //integrate light contribution Back to Front.
  if(x<=resolution.x && y<=resolution.y){
    int vert = 0;
    if(eyePaths[index].vert[vert].isValid){
        float weight = imageWeights[index];
        float solidAngle = eyePaths[index].vert[vert].solidAngle;
        float denom  = weight + solidAngle;
        colors[index] = colors[index] * (weight/denom) + eyePaths[index].vert[vert].directLight * (solidAngle /denom);
        imageWeights[index] = denom;
        return;
      }
  }
}


__global__ void MISRenderColor(glm::vec2 resolution, glm::vec3* colors, float* imageWeights, int traceDepth, Path* eyePaths, float time, staticGeom* geoms, 
	int numberOfGeoms, material* materials) {
  // index into array is based off pixel position
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  //integrate light contribution Back to Front.
  if(x<=resolution.x && y<=resolution.y){
    glm::vec3 inDirection;
    glm::vec3 outDirection;
    glm::vec3 normal;
    ray thisRay;
    glm::vec3 BSDFcolor = glm::vec3(0);
    glm::vec3 inColor = glm::vec3(0);
    float solidAngle;
    float pdfWeight;
    glm::vec3 directLight;
    
    int validRay = 0;
    float totalPDFWeight = 0.0f;
    vertex v;
    int max = traceDepth - 1;
    for(int vert = max ; vert >= 0; vert--){
      v = eyePaths[index].vert[vert];
      if(v.isValid == 1){
        validRay = 1;
        material mat = v.mat;
        if(v.hitLight == 1){
          //This vertex is on a light
          inColor = mat.color * mat.emittance;
          totalPDFWeight = 1.0f;
        }else{
          totalPDFWeight *= v.pdfWeight;
          //update BSDF color
          inDirection  = v.inDirection;
          outDirection = v.outDirection;
          normal       = v.normal;
          pdfWeight    = v.pdfWeight;
          BSDFcolor    = getColorFromBSDF(inDirection, outDirection, normal, inColor, mat);
         inColor = BSDFcolor;
          /*
          //update incoming color
          solidAngle  = v.solidAngle;
          directLight = v.directLight;
          
          //power heuristic
          pdfWeight  *= pdfWeight;
          solidAngle *= solidAngle;
          
          // balance heuristic to update incolor
          float denom = solidAngle + pdfWeight;
          //inColor     = (solidAngle/denom) * directLight + (pdfWeight/denom) * BSDFcolor;
          inColor     = directLight + BSDFcolor;
          */
        }
      }
    }
    if(validRay == 1){
      //Update Pixel Color
    	/*
      float weight = imageWeights[index];
      float denom  = weight + totalPDFWeight;
      colors[index] = colors[index] * (weight/denom) + inColor * (totalPDFWeight/denom);
      imageWeights[index] = denom;
      */
    	v = eyePaths[index].vert[0];
    	pdfWeight = totalPDFWeight;
    	 solidAngle = v.solidAngle;

		 //pdfWeight *= pdfWeight;
		 //solidAngle *= solidAngle;

		 float denom = pdfWeight + solidAngle;
		 inColor = inColor * (pdfWeight/denom) + v.directLight * (solidAngle/denom);
		 float weight = imageWeights[index];
		 denom  = weight + 1.0f;
		 //debug:
		 //inColor = glm::vec3(pdfWeight);

		 colors[index] = colors[index] * (weight/denom) + inColor * (1.0f/denom);
		 imageWeights[index] = denom;
    }
  }
}

__global__ void BiDirRenderColor(glm::vec2 resolution, glm::vec3* colors, float* imageWeights, int traceDepth, Path* eyePaths, float time, staticGeom* geoms,
	int numberOfGeoms, material* materials, Path* lightPaths) {
  // index into array is based off pixel position

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  //integrate light contribution Back to Front.
  if(x<=resolution.x && y<=resolution.y){
	glm::vec3 inDirection;
    glm::vec3 outDirection;
    glm::vec3 normal;
    glm::vec3 averageBSDF;
    float sumPDF = 0.0f;
    float numPDFs = 0.0f;
    ray thisRay;
    glm::vec3 BSDFcolor = glm::vec3(0);
    glm::vec3 inColor;
    float solidAngle;
    float pdfWeight;
    glm::vec3 directLight;
    int validRay = 0;
    float totalPDFWeight = 1.0f;
    material mat;
    vertex v;

    //for each potential light path calculate
    // - Incoming Light (flux retained from light)
    // - PDFWeight (accumulated)
    // - position of final vertex
    for(int LP = 0; LP <= 4; LP++){
		//int LP = 2;
    	vertex lv = lightPaths[LP].vert[0];
		glm::vec3 lightColor = lv.mat.color * lv.mat.emittance;
		float lightPDF = 1.0f;
		for(int li = 1; li <= traceDepth; li ++){
			//connect light vertex to each eye vertex
			for( int max = traceDepth - 1; max >=0; max--){
				totalPDFWeight = lightPDF;
				inColor = lightColor;
				validRay = 0;
				v = eyePaths[index].vert[max];
				//check if can connect.  if no break
				 if(v.isValid == 1){
				//intersection checks:
					float dist = glm::distance(lv.position, v.position);
					float distToIntersect = dist;
					thisRay.direction = glm::normalize(lv.position - v.position);
					thisRay.origin = v.position;

					int materialID = 0;                        //updated in intersectionTest
					glm::vec3 intersectPoint, intersectNormal; //updated in intersectionTest
					distToIntersect = intersectionTest(geoms, numberOfGeoms, materialID, thisRay, distToIntersect, intersectNormal, intersectPoint);
					if(distToIntersect < dist){//hit something
						material m = materials[materialID];
						if(m.type != 9){
							continue;
						}
					}
					//update inColor
					inColor = getColorFromBSDF(v.inDirection, thisRay.direction , v.normal, inColor, v.mat);
					//update Probability
					pdfWeight = PDF(v.inDirection, thisRay.direction, v.normal, mat);
					totalPDFWeight *= pdfWeight;
					for(int vert = max - 1 ; vert >= 0; vert--){
					  v = eyePaths[index].vert[vert];
					  if(v.isValid == 1){
						validRay = 1;
						material mat = v.mat;
						if(v.hitLight == 1){
							inColor = mat.color * mat.emittance;
							totalPDFWeight = 1.0f;//*= PDF(v.inDirection, v.outDirection, v.normal, v.mat);
						}else{
						  totalPDFWeight *= v.pdfWeight;
						  //update BSDF color
						  BSDFcolor    = getColorFromBSDF(v.inDirection, v.outDirection, v.normal, inColor, v.mat);
						  //inColor     = v.directLight + BSDFcolor;
						  inColor = BSDFcolor;
						  /*
						   BSDFcolor    = getColorFromBSDF(v.inDirection, v.outDirection, v.normal, inColor, v.mat);

						   //power heuristic
						   pdfWeight  = v.pdfWeight * v.pdfWeight;
						   solidAngle = v.solidAngle * v.solidAngle;

						   // balance heuristic to update incolor
						   float denom = solidAngle + pdfWeight;
						   inColor     = (solidAngle/denom) * v.directLight + (pdfWeight/denom) * BSDFcolor;
						   //inColor     = directLight + BSDFcolor;
							*/
						  //inColor      = getColorFromBSDF(v.inDirection, v.outDirection, v.normal, inColor, v.mat);

						}
					  }
					}
					if(validRay == 1 && totalPDFWeight > 0){
/*
						//Update Pixel Color
						float weight = imageWeights[index];
						float denom  = weight + totalPDFWeight;
						//float MISDenom = eyePaths[index].vert[0].pdfWeight + eyePaths[index].vert[0].solidAngle;
						colors[index] = colors[index] * (weight/denom) + inColor * (totalPDFWeight/denom);
						imageWeights[index] = denom;
*/
						float weight = sumPDF;
						float denom = sumPDF + totalPDFWeight;
						averageBSDF = averageBSDF * (weight/denom) + inColor * (totalPDFWeight/denom);
						sumPDF = denom;
						numPDFs += 1.0f;
					}
				 }
				}
			// Update light vertex
			if(li < traceDepth){
				lv = lightPaths[LP].vert[li];
				if(lv.isValid == 0){
					break; //unless we want to do the second pass on eye paths below
				}
				lightColor = getColorFromBSDF(lv.inDirection, lv.outDirection, lv.normal, lightColor, lv.mat);
				lightPDF *= lv.pdfWeight;
			}
		}
    }
    //MIS HEURISTIC WITH DIRECT LIGHT!
	v = eyePaths[index].vert[0];//direct light first bounce
	//power heuristic
	if(numPDFs > 0.0){
		pdfWeight  = sumPDF/numPDFs;
	}else{
		pdfWeight = 0.0;
	}
	if(v.isValid == 1){

		 solidAngle = v.solidAngle;

		 pdfWeight *= pdfWeight;
		 solidAngle *= solidAngle;

		 float denom = pdfWeight + solidAngle;
		 inColor = averageBSDF * (pdfWeight/denom) + v.directLight * (solidAngle/denom);
		 float weight = imageWeights[index];
		 denom  = weight + 1.0f;
		 //debug:
		 //inColor = glm::vec3(pdfWeight);

		 colors[index] = colors[index] * (weight/denom) + inColor * (1.0f/denom);
		 imageWeights[index] = denom;


	}else if (pdfWeight > 0.0f){
		float weight = imageWeights[index];
		float denom  = weight + 1.0f;
		colors[index] = colors[index] * (weight/denom) + averageBSDF * (1.0f/denom);
		imageWeights[index] = denom;
	}
  }
}

__global__ void compactRays(int* scanRays, rayState* rayList, int* validRays, int length){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index >= length){
    return;
  }
  validRays[index] = 0;
  if(index == 0){//first 
    return;
  }

  if(scanRays[index - 1] < scanRays[index]){
    rayState newRay = rayList[index];
    __syncthreads();
    rayList[scanRays[index]] = newRay;
    validRays[scanRays[index]] = 1;
  }
}


// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms, geom* lights, int numberOfLights, int renderType){
  
  int traceDepth = 4; //determines how many bounces the raytracer traces

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  // send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage,           (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  // allocate eye path per pixel
  Path* eyePaths = NULL;
  hipMalloc((void**)&eyePaths,           (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(Path));

  // allocate Light paths
	Path* lightPaths = NULL;
	//hipMalloc((void**)&lightPaths,         (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(Path));
	hipMalloc((void**)&lightPaths,         10 * sizeof(Path));

  // Allocate per-pixel accumulated weight (probabilites of valid light paths)
  float* imageWeights = NULL;
  hipMalloc((void**)&imageWeights,                  (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(float));
  hipMemcpy( imageWeights, renderCam->imageWeights, (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(float), hipMemcpyHostToDevice);
  
  // package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;
  }
  
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms,   numberOfGeoms * sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms * sizeof(staticGeom), hipMemcpyHostToDevice);
  
  
  // package Lights and sent to GPU
  staticGeom* lightList = new staticGeom[numberOfLights];
  for(int i=0; i<numberOfLights; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = lights[i].type;
    newStaticGeom.materialid = lights[i].materialid;
    newStaticGeom.translation = lights[i].translations[frame];
    newStaticGeom.rotation = lights[i].rotations[frame];
    newStaticGeom.scale = lights[i].scales[frame];
    newStaticGeom.transform = lights[i].transforms[frame];
    newStaticGeom.inverseTransform = lights[i].inverseTransforms[frame];
    lightList[i] = newStaticGeom;
  }
  
  staticGeom* cudalights = NULL;
  hipMalloc((void**)&cudalights,    numberOfLights * sizeof(staticGeom));
  hipMemcpy( cudalights, lightList, numberOfLights * sizeof(staticGeom), hipMemcpyHostToDevice);
  
  
  // package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;
  cam.DOF = renderCam->DOF[frame];//new
  cam.APERATURE = renderCam->APERATURE[frame];//new
  
  // package materials
  material* materialList = NULL;
  hipMalloc((void**) &materialList,   numberOfMaterials * sizeof(material));
  hipMemcpy( materialList, materials, numberOfMaterials * sizeof(material), hipMemcpyHostToDevice);
  
  //allocate Rays
  rayState* rayList = NULL;
  hipMalloc((void**)&rayList, (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(rayState));


  //allocate light rays 
  rayState* lightrayList = NULL;
  //hipMalloc((void**)&lightrayList, (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(rayState));
  hipMalloc((void**)&lightrayList, 10 * sizeof(rayState));
  
 // kernel launches
  //Get initial rays
  initializeRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, rayList);

  //Initialize light subpaths
  int numLightpaths = 10;

  initializeLightPaths<<<1, numLightpaths>>>((float)iterations, cam, lightrayList, numLightpaths,  cudalights, numberOfLights, lightPaths,  materialList);
  

  //build eye path
  for(int i = 0; i < traceDepth; i++){
    //do one step
    buildEyePath<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms, materialList, numberOfMaterials, rayList, i, eyePaths, cudalights, numberOfLights);
  }

/*
  // error checking
  Path* localPaths = new Path[(int)renderCam->resolution.x * (int)renderCam->resolution.y];
  hipMemcpy( localPaths, eyePaths, (int)renderCam->resolution.x * (int)renderCam->resolution.y * sizeof(Path), hipMemcpyDeviceToHost);
  for(int i = 0; i < (int)renderCam->resolution.x * (int)renderCam->resolution.y; i++){
	  vertex v = localPaths[i].vert[1];
	  if(v.isValid == 1){
		  if(v.solidAngle < 0.0 || v.solidAngle > 1.0){
			  std::cout << "V" << i << " solid angle: " << v.solidAngle << std::endl;
			  std::cout << "V" << i << " pdfWeight: " << v.pdfWeight << std::endl;
		  }else if (v.pdfWeight < 0.0 || v.pdfWeight > 1.0){
			  //std::cout << "V" << i << " solid angle: " << v.solidAngle << std::endl;
			  std::cout << "V" << i << " pdfWeight: " << v.pdfWeight << std::endl;
		  }
	  }
  }
  exit(0);
*/

  //build light path
    for(int i = 1; i < traceDepth; i++){
      //do one step
      buildLightPath<<<1, numLightpaths>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms, materialList, numberOfMaterials, lightrayList, i, lightPaths, cudalights, numberOfLights);
    }
/*
   //buildLightPath
  for(int i = 0; i < traceDepth; i++){
    //do one step
    buildEyePath<<<1, numLightpaths>>>(glm::vec2(10,1), (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms, materialList, numberOfMaterials, lightrayList, i, lightPaths);
  }
*/


/*  
  //connect paths and render to screen
  connectPaths<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, cudaimage, imageWeights, cudageoms, numberOfGeoms, traceDepth, eyePaths, lightPaths);
*/
if(renderType == 0){//classic PathTracer
  RenderColor<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, cudaimage, imageWeights, traceDepth, eyePaths);
}else if(renderType == 1){ //Direct Lighting Only
  RenderDirectLight<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, cudaimage, imageWeights, traceDepth, eyePaths);
}else if(renderType == 2){//Multiple Importance Sampling
  MISRenderColor<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, cudaimage, imageWeights, traceDepth, eyePaths, (float)iterations, cudageoms, numberOfGeoms, materialList);
}else{
	BiDirRenderColor<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, cudaimage, imageWeights, traceDepth, eyePaths, (float)iterations, cudageoms, numberOfGeoms, materialList, lightPaths);
}
  //update visual
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);

  // retrieve image from GPU
  hipMemcpy( renderCam->image,        cudaimage,    (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);
  //retrieve weights from GPU
  hipMemcpy( renderCam->imageWeights, imageWeights, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(float), hipMemcpyDeviceToHost);
/*
  //debug
  for(int i = 0; i < (int)renderCam->resolution.x * (int)renderCam->resolution.y; i++){
	  float weight = renderCam->imageWeights[i];
	  if(weight == 0.0 || weight == 1.0){
		  std::cout << "pixel: " << i << " weight: " << weight << std::endl;
	  }
  }
  exit(0);
*/

  // free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  hipFree( cudalights ); //added
  hipFree(materialList); //added
  hipFree(rayList);      //added
  hipFree(lightrayList); //VCM added
  hipFree(eyePaths);     //added
  hipFree(lightPaths);     //added
  hipFree(imageWeights); //added
  delete geomList;
  delete lightList;//ADDED

  // make certain the kernel has completed
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}

